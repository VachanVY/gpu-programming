#include "hip/hip_runtime.h"
// As is from https://github.com/Maharshi-Pandya/cudacodes/blob/master/query-device/main.cu

#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>
#include <stdio.h>

int main() {
    int dev_count;
    hipDeviceProp_t prop;

    hipGetDeviceCount(&dev_count);
    hipGetDeviceProperties(&prop, 0);

    printf(">> CUDA enabled devices in the system: %d\n", dev_count);
    printf(">> Compute capability: %d.%d\n", prop.major, prop.minor);

    printf(">> Max grid size: (%d, %d, %d)\n", prop.maxGridSize[0], prop.maxGridSize[1], prop.maxGridSize[2]);
    printf(">> Max block size: %d\n", prop.maxThreadsPerBlock);

    printf(">> Number of SMs: %d\n", prop.multiProcessorCount);
    printf(">> Clock rate of the SMs (in kHz): %d\n", prop.clockRate);

    printf(">> Max threads dimension: (%d, %d, %d)\n", prop.maxThreadsDim[0], prop.maxThreadsDim[1], prop.maxThreadsDim[2]);
    printf(">> Max threads per SM: %d\n", prop.maxThreadsPerMultiProcessor);

    printf(">> Registers available per block: %d\n", prop.regsPerBlock);
    printf(">> Registers available per SM: %d\n", prop.regsPerMultiprocessor);

    printf(">> Warp size (threads per warp): %d\n", prop.warpSize);
    printf(">> Shared memory size per block: %zd bytes\n", prop.sharedMemPerBlock);
    printf(">> Shared memory size per SM: %zd bytes\n", prop.sharedMemPerMultiprocessor);

    printf(">> L2 cache size: %d bytes\n", prop.l2CacheSize);

    printf(">> Memory bus width: %d bits\n", prop.memoryBusWidth);
    printf(">> Memory clock rate: %d KHz\n", prop.memoryClockRate);

    int cudaCores = prop.multiProcessorCount * 128;
    float clockGHz = prop.clockRate / 1e6;
    float gflops = cudaCores * clockGHz * 2;

    printf(">> Theoretical Max GFLOPS: %.2f\n", gflops);

    float memoryBandwidth = (2 * prop.memoryClockRate * prop.memoryBusWidth) / (8.0 * 1e6);
    printf(">> Maximum Memory Bandwidth: %.2f GB/s\n", memoryBandwidth);
}

// >> CUDA enabled devices in the system: 1
// >> Compute capability: 8.9
// >> Max grid size: (2147483647, 65535, 65535)
// >> Max block size: 1024
// >> Number of SMs: 128
// >> Clock rate of the SMs (in kHz): 2550000
// >> Max threads dimension: (1024, 1024, 64)
// >> Max threads per SM: 1536
// >> Registers available per block: 65536
// >> Registers available per SM: 65536
// >> Warp size (threads per warp): 32
// >> Shared memory size per block: 49152 bytes
// >> Shared memory size per SM: 102400 bytes
// >> L2 cache size: 75497472 bytes
// >> Memory bus width: 384 bits
// >> Memory clock rate: 10501000 KHz
// >> Theoretical Max GFLOPS: 83558.40
// >> Maximum Memory Bandwidth: -65.65 GB/s